/*#include <iostream>
#include <cuda.h>

using namespace std ;



__global__ void hadamard(float *a, float*b, float* res){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        red[id] = a[id] * b[id];
}
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
 
// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}
 

